#include "hip/hip_runtime.h"
#include <sys/types.h>
#include <sys/socket.h>
#include <netinet/in.h>
#include <string.h>
#include <unistd.h>
#include <sys/uio.h>
#include <errno.h>
#include <stdio.h>
#include <math.h>
#include <pthread.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>

#define NUM_BLOCKS 4096
#define THREADS_PER_BLOCK 128

#define CELL_DIMX 3.0f
#define CELL_DIMY 3.0f
#define CELL_DIMZ 3.0f
#define CELL_NUMX 256
#define CELL_NUMY 256
#define CELL_NUMZ 256

#define MAXX (CELL_NUMX*CELL_DIMX)
#define MAXY (CELL_NUMY*CELL_DIMY)
#define MAXZ (CELL_NUMZ*CELL_DIMZ)

#define EPSILON 0.01f

#define MAXXE (MAXX-EPSILON)
#define MAXYE (MAXY-EPSILON)
#define MAXZE (MAXZ-EPSILON)

#define H_CONSTANT 1.5f
#define H_CONSTANT_TIMES_2 (2.0f*H_CONSTANT)
#define H_CONSTANT_TIMES_2_SQUARED (4.0f*H_CONSTANT*H_CONSTANT)
#define W_CONSTANT 15.0f/(16.0f*PI*H_CONSTANT*H_CONSTANT*H_CONSTANT)

#define VISC 1.0f					// This corresponds to MU_N
#define VISC_BASE 1.0f				// This corresponds to MU_X - MU_N
#define P_0 1.0f					// This is RHO_0 times c squared

#define BOUNDARY_THRESH 0.20f		// This corresponds to BETA

#define GRAVITY_CONSTANT -0.00005f	// This corresponds to g

// The variables KAPPA, u, w and q from the paper are stored in a single cuda float4 type, so here they
// are referred to has x,y,z and w respectively.

#define RHO_0 1.0f

#define BOUNDARY_VISC_CONSTANT 10.0f	// This corresponds to T

#define PI 3.14159265358979323846264f

__device__ float LengthVector(float3 a)
{
	return sqrtf(a.x*a.x+a.y*a.y+a.z*a.z);
}

__device__ float LengthVector(float4 a)
{
	return sqrtf(a.x*a.x+a.y*a.y+a.z*a.z);
}

__device__ float3 MultiplyVector(float3 a, float f)
{
	return make_float3(a.x*f,a.y*f,a.z*f);
}

__device__ float WCalcGivenS(float s)
{
	if (s>=2.0f)
	{
		return 0.0f;
	}
	else
	{
		return W_CONSTANT * (0.25f*s*s-s+1.0f);
	}
}

__device__ float WCalc(float3 rij,float dist)
{
	float s = dist/H_CONSTANT;

	if (s>=2.0f)
	{
		return 0.0f;
	}
	else
	{
		return W_CONSTANT * (0.25f*s*s-s+1.0f);
	}
}

__device__ float3 DeltaWCalc(float3 rij,float dist)
{
	float s = dist/H_CONSTANT;

	if (s>=2.0f)
	{
		return make_float3(0.0f,0.0f,0.0f);
	}
	else
	{
		return MultiplyVector(rij,W_CONSTANT*(0.5f*s-1.0f)/(H_CONSTANT*dist));
	}
}

// This is the part of DeltaWCalc that multiplies the vector
__device__ float FabCalc(float dist)
{
	float s = dist/H_CONSTANT;

	if (s>=2.0f)
	{
		return 0.0f;
	}
	else
	{
		return W_CONSTANT*(0.5f*s-1.0f)/(H_CONSTANT*dist);
	}
}

__device__ float DotProduct(float3 a, float3 b)
{
	return a.x*b.x+a.y*b.y+a.z*b.z;
}

// A sigmoid curve that will go from y near 0 to y near 1 as x goes from 0 to 1
__device__ float Sigmoid(float x)
{
	return 1.0f/(1.0f+expf(-12.0f*(x-0.5f)));
}

__device__ float BoundaryViscForce(float y)
{
	return BOUNDARY_VISC_CONSTANT*(0.5f-y)*(0.5f-y);
}

__global__ void InitialiseDevice(float4 *pPos, float4 *pAcc, unsigned *cellHash, unsigned *pIndex, unsigned *trackIndex, float *d_simTime, int nParticles, int nloops)
{
	int thdsPerBlk = blockDim.x;	// Number of threads per block
	int blkIdx = blockIdx.x;		// Index of block
	int thdIdx = threadIdx.x;		// Index of thread within a block
	int zIdx = blkIdx*thdsPerBlk*nloops + thdIdx;
	int stepSize = blockDim.x*gridDim.x;

	for(int i = 0; i<nloops && zIdx < nParticles; i++)
	{
		if (zIdx == 0)
		{
			*d_simTime = 0.0f;
		}

		cellHash[zIdx] = 
			(((int)(pPos[zIdx].x/CELL_DIMX))*CELL_NUMX +
			 ((int)(pPos[zIdx].y/CELL_DIMY)))*CELL_NUMY +
		          (int)(pPos[zIdx].z/CELL_DIMZ);
		pIndex[zIdx] = zIdx;
		trackIndex[zIdx] = zIdx;
		pAcc[zIdx] = make_float4(0.0f,0.0f,0.0f,0.0f);

		zIdx += stepSize;
	}
}

__global__ void InitCellStart(int *cellStart, int nloops)
{
	int thdsPerBlk = blockDim.x;	// Number of threads per block
	int blkIdx = blockIdx.x;		// Index of block
	int thdIdx = threadIdx.x;		// Index of thread within a block
	int zIdx = blkIdx*thdsPerBlk*nloops + thdIdx;
	int stepSize = blockDim.x*gridDim.x;
	
	for(int i = 0; i<nloops && zIdx < CELL_NUMX*CELL_NUMY*CELL_NUMZ; i++)
	{
		cellStart[zIdx] = -1;

		zIdx += stepSize;
	}
}

__global__ void ArrayCopy(float4 *a0, float4 *a1, float4 *b0, float4 *b1, unsigned *cellHash, int *cellStart, unsigned *pIndex, int nParticles, int nloops)
{
	int thdsPerBlk = blockDim.x;	// Number of threads per block
	int blkIdx = blockIdx.x;		// Index of block
	int thdIdx = threadIdx.x;		// Index of thread within a block
	int zIdx = blkIdx*thdsPerBlk*nloops + thdIdx;
	int stepSize = blockDim.x*gridDim.x;

	for(int i = 0; i<nloops && zIdx < nParticles; i++)
	{
		unsigned p = pIndex[zIdx];
		a1[zIdx] = a0[p];
		b1[zIdx] = b0[p];

		if (zIdx == 0 || cellHash[zIdx] != cellHash[zIdx-1])
		{
			cellStart[cellHash[zIdx]] = zIdx;
		}

		zIdx += stepSize;
	}
}

__global__ void UpdateTrackIndex(unsigned *pIndex, unsigned *trackIndex0, unsigned *trackIndex1, int nParticles, int nloops)
{
	int thdsPerBlk = blockDim.x;	// Number of threads per block
	int blkIdx = blockIdx.x;		// Index of block
	int thdIdx = threadIdx.x;		// Index of thread within a block
	int zIdx = blkIdx*thdsPerBlk*nloops + thdIdx;
	int stepSize = blockDim.x*gridDim.x;

	for(int i = 0; i<nloops && zIdx < nParticles; i++)
	{
		unsigned p = pIndex[zIdx];
		trackIndex1[zIdx] = trackIndex0[p];

		zIdx += stepSize;
	}
}

__global__ void ParticleMove(float4 *pPos, float4 *pVel, float4 *pAcc, int *pBoundary, unsigned *cellHash, unsigned *pIndex, float *maxAcc, float *simTime, int nParticles, int nloops)
{
	int thdsPerBlk = blockDim.x;	// Number of threads per block
	int blkIdx = blockIdx.x;		// Index of block
	int thdIdx = threadIdx.x;		// Index of thread within a block
	int zIdx = blkIdx*thdsPerBlk*nloops + thdIdx;
	int stepSize = blockDim.x*gridDim.x;

	float deltaT = 0.25f*sqrtf(H_CONSTANT/(*maxAcc));

	if (deltaT>0.05f) deltaT = 0.05f;

	if (zIdx == 0)
	{
		*simTime += deltaT;
	}

	for(int i = 0; i<nloops && zIdx < nParticles; i++)
	{
		pPos[zIdx].x += (pVel[zIdx].x + pAcc[zIdx].x*deltaT)*deltaT;
		pPos[zIdx].y += (pVel[zIdx].y + pAcc[zIdx].y*deltaT)*deltaT;
		pPos[zIdx].z += (pVel[zIdx].z + pAcc[zIdx].z*deltaT)*deltaT;

		pVel[zIdx].x += pAcc[zIdx].x*deltaT;
		pVel[zIdx].y += pAcc[zIdx].y*deltaT;
		pVel[zIdx].z += pAcc[zIdx].z*deltaT;
		pVel[zIdx].w += pAcc[zIdx].w*deltaT;

		// Apply gravitational force at this point, ready for next iteration
		// pAcc[zIdx].w is deltaDensity - set it to zero here
		pAcc[zIdx] = make_float4(0.0f,GRAVITY_CONSTANT,0.0f,0.0f);

		if (pPos[zIdx].x >= MAXXE) {pPos[zIdx].x = MAXXE; pVel[zIdx].x = 0.0f;}
		if (pPos[zIdx].y >= MAXYE) {pPos[zIdx].y = MAXYE; pVel[zIdx].y = 0.0f;}
		if (pPos[zIdx].z >= MAXZE) {pPos[zIdx].z = MAXZE; pVel[zIdx].z = 0.0f;}
		if (pPos[zIdx].x < 0.0f) {pPos[zIdx].x = 0.0f; pVel[zIdx].x = 0.0f;}
		if (pPos[zIdx].y < 0.0f)
		{
			pPos[zIdx].y = 0.0f;
			pVel[zIdx].y = 0.0f;
		}
		if (pPos[zIdx].y < 0.5f) 
		{
			pVel[zIdx].y *= (1.0f - BoundaryViscForce(pPos[zIdx].y)*deltaT);
			pVel[zIdx].x *= (1.0f - BoundaryViscForce(pPos[zIdx].y)*deltaT);
			pVel[zIdx].z *= (1.0f - BoundaryViscForce(pPos[zIdx].y)*deltaT);
		}

		if (pPos[zIdx].z < 0.0f) {pPos[zIdx].z = 0.0f; pVel[zIdx].z = 0.0f;}

		cellHash[zIdx] = 
			(((int)(pPos[zIdx].x/CELL_DIMX))*CELL_NUMX +
			 ((int)(pPos[zIdx].y/CELL_DIMY)))*CELL_NUMY +
		          (int)(pPos[zIdx].z/CELL_DIMZ);

		pIndex[zIdx] = zIdx;

		zIdx += stepSize;
	}
}

#define CELLNUMAUX(x,y,z) ((x)*CELL_NUMX+(y))*CELL_NUMY+(z)
#define CELLNUM(x,y,z) ((x)<0 || (y)<0 || (z)<0 || (x)>=CELL_NUMX || (y)>=CELL_NUMY || (z)>=CELL_NUMZ)?-1:CELLNUMAUX(x,y,z)

__global__ void ShepardFilter(float4 *pPos, float4 *pVel, float *pNewDensity, unsigned *cellHash, int *cellStart, int nParticles, int nloops)
{
	int thdsPerBlk = blockDim.x;	// Number of threads per block
	int blkIdx = blockIdx.x;		// Index of block
	int thdIdx = threadIdx.x;		// Index of thread within a block
	int zIdx = blkIdx*thdsPerBlk*nloops + thdIdx;
	int stepSize = blockDim.x*gridDim.x;

	int neighbourCell,ps;
	int cellx,celly,cellz;

	float numerator;
	float denominator;

	for(int i = 0; i<nloops && zIdx < nParticles; i++)
	{
		cellx = (int)(pPos[zIdx].x/CELL_DIMX); 
		celly = (int)(pPos[zIdx].y/CELL_DIMY);
		cellz = (int)(pPos[zIdx].z/CELL_DIMZ);

		numerator = W_CONSTANT;
		denominator = W_CONSTANT/pVel[zIdx].w;

		for(int xo=-1;xo<=1;xo++) for(int yo=-1;yo<=1;yo++) for(int zo=-1;zo<=1;zo++)
		{	
			if ((neighbourCell = CELLNUM(cellx+xo,celly+yo,cellz+zo)) != -1 &&
			    (ps = cellStart[neighbourCell]) != -1)
			{
				while(ps < nParticles && cellHash[ps] == neighbourCell)
				{
					if (ps != zIdx)
					{
						float3 diff = make_float3(
					        	pPos[ps].x - pPos[zIdx].x,
							pPos[ps].y - pPos[zIdx].y,
							pPos[ps].z - pPos[zIdx].z);

						float dist2 = diff.x*diff.x+diff.y*diff.y+diff.z*diff.z;
						if (dist2 < H_CONSTANT_TIMES_2_SQUARED)
						{
							float dist = sqrtf(dist2);
							float w = WCalc(diff,dist);

							numerator += w;
							denominator += w/pVel[ps].w;
						}
					}
					ps++;
				}
			}
		}

		pNewDensity[zIdx] = numerator/denominator;

		zIdx += stepSize;
	}
}

__global__ void UpdateDensity(float4 *pVel, float *pNewDensity, int nParticles, int nloops)
{
	int thdsPerBlk = blockDim.x;	// Number of threads per block
	int blkIdx = blockIdx.x;		// Index of block
	int thdIdx = threadIdx.x;		// Index of thread within a block
	int zIdx = blkIdx*thdsPerBlk*nloops + thdIdx;
	int stepSize = blockDim.x*gridDim.x;

	for(int i = 0; i<nloops && zIdx < nParticles; i++)
	{
		pVel[zIdx].w = pNewDensity[zIdx];

		zIdx += stepSize;
	}
}

__global__ void ParticleForces(float4 *pPos, float4 *pVel, float4 *pAcc, float *pAccLength, int *pBoundary, unsigned *cellHash, int *cellStart, int nParticles, int nloops, unsigned *trackIndex)
{
	int thdsPerBlk = blockDim.x;	// Number of threads per block
	int blkIdx = blockIdx.x;		// Index of block
	int thdIdx = threadIdx.x;		// Index of thread within a block
	int zIdx = blkIdx*thdsPerBlk*nloops + thdIdx;
	int stepSize = blockDim.x*gridDim.x;

	int neighbourCell,ps;
	int cellx,celly,cellz;
	
	float4 thisAcc;
	float thisViscosity;
	float4 vThis;
	float3 boundaryVector;
	float boundaryDensity;

	for(int i = 0; i<nloops && zIdx < nParticles; i++)
	{
		cellx = (int)(pPos[zIdx].x/CELL_DIMX); 
		celly = (int)(pPos[zIdx].y/CELL_DIMY);
		cellz = (int)(pPos[zIdx].z/CELL_DIMZ);

		vThis = pVel[zIdx];

		thisAcc = make_float4(0.0f,0.0f,0.0f,0.0f);
		thisViscosity = pPos[zIdx].w;

		boundaryVector = make_float3(0.0f,0.0f,0.0f);
		boundaryDensity = 0.0f;

		for(int xo=-1;xo<=1;xo++) for(int yo=-1;yo<=1;yo++) for(int zo=-1;zo<=1;zo++)
		{	
			if ((neighbourCell = CELLNUM(cellx+xo,celly+yo,cellz+zo)) != -1 &&
			    (ps = cellStart[neighbourCell]) != -1)
			{
				while(ps < nParticles && cellHash[ps] == neighbourCell)
				{
					if (ps != zIdx)
					{
						float3 diff = make_float3(
					        	pPos[zIdx].x - pPos[ps].x,
							pPos[zIdx].y - pPos[ps].y,
							pPos[zIdx].z - pPos[ps].z);

						float dist2 = diff.x*diff.x + diff.y*diff.y + diff.z*diff.z;
						if (dist2 < H_CONSTANT_TIMES_2_SQUARED)
						{
						    float dOther = pVel[ps].w;
						    float dist = sqrtf(dist2);
						    float3 dw = DeltaWCalc(diff,dist);
						    float fab = FabCalc(dist);
						    float w = WCalc(diff,dist);
						    float3 vdiff = make_float3(
						    	vThis.x - pVel[ps].x,
						    	vThis.y - pVel[ps].y,
						    	vThis.z - pVel[ps].z);

						    // Viscous forces
						    if (1)
						    {
							float a = (thisViscosity+pPos[ps].w)/(vThis.w*dOther*dist2);
							a *= DotProduct(diff,dw);
							thisAcc.x += vdiff.x * a;
							thisAcc.y += vdiff.y * a;
							thisAcc.z += vdiff.z * a;
						    }
						    // Pressure forces
						    if (1)
						    {
							float pterm = P_0 *
							 (1.0f/(RHO_0*vThis.w)-1.0f/(vThis.w*vThis.w)
							+ 1.0f/(RHO_0*dOther)-1.0f/(dOther*dOther));
							thisAcc.x -= dw.x*pterm;
							thisAcc.y -= dw.y*pterm;
							thisAcc.z -= dw.z*pterm;
						    }
						    // Density change
						    if (1)
						    {
							    thisAcc.w += DotProduct(vdiff,dw);
						    }
						    // Boundary vector
						    if (1)
						    {
							    boundaryVector.x += diff.x*w;
							    boundaryVector.y += diff.y*w;
							    boundaryVector.z += diff.z*w;
							    boundaryDensity += w;
						    }						    
						}
					}
					ps++;
				}
			}
		}

		pAcc[zIdx].x += thisAcc.x;
		pAcc[zIdx].y += thisAcc.y;
		pAcc[zIdx].z += thisAcc.z;
		pAcc[zIdx].w += thisAcc.w;

		pBoundary[zIdx] = boundaryDensity < 0.00001f || LengthVector(boundaryVector)/boundaryDensity > BOUNDARY_THRESH;

		pAccLength[zIdx] = LengthVector(pAcc[zIdx]);

		zIdx += stepSize;
	}
}

int nParticles;
float4 *d_pVel[2];
float *d_pNewDensity;
float4 *d_pPos[2];
float4 *d_pAcc;
float *d_pAccLength;
int *d_pBoundary;
unsigned *d_cellHash;
int *d_cellStart;
unsigned *d_pIndex;
unsigned *d_trackIndex[2];
float *d_simTime;

float4 *h_pVel;
float4 *h_pPos;
float h_simTime;
unsigned *h_trackIndex;

int activeArray;

void Check(hipError_t e)
{
	if (e != hipSuccess)
	{
		printf("%s\n",hipGetErrorString(hipGetLastError()));
		exit(-1);
	}
}

float RandFloat(float min, float max)
{
	return min + (max-min)*((rand()%10000)/10000.0f);
}

void CopyToDevice(void)
{
	hipMemcpy(d_pVel[activeArray],h_pVel,sizeof(float4)*nParticles,hipMemcpyHostToDevice);
	hipMemcpy(d_pPos[activeArray],h_pPos,sizeof(float4)*nParticles,hipMemcpyHostToDevice);
}

void CopyFromDevice(void)
{
	hipMemcpy(h_pVel,d_pVel[activeArray],sizeof(float4)*nParticles,hipMemcpyDeviceToHost);
	hipMemcpy(h_pPos,d_pPos[activeArray],sizeof(float4)*nParticles,hipMemcpyDeviceToHost);
	hipMemcpy(h_trackIndex,d_trackIndex[activeArray],sizeof(unsigned)*nParticles,hipMemcpyDeviceToHost);
	hipMemcpy(&h_simTime,d_simTime,sizeof(float),hipMemcpyDeviceToHost);
}

void AllocateMemory(void)
{
	for(int i = 0; i<2; i++)
	{
		Check( hipMalloc((void**)&d_pVel[i],sizeof(float4)*nParticles) );
		Check( hipMalloc((void**)&d_pPos[i],sizeof(float4)*nParticles) );
		Check( hipMalloc((void**)&d_trackIndex[i],sizeof(unsigned)*nParticles) );
	}
	Check( hipMalloc((void**)&d_pNewDensity,sizeof(float)*nParticles) );
	Check( hipMalloc((void**)&d_pBoundary,sizeof(int)*nParticles) );
	Check( hipMalloc((void**)&d_pAcc,sizeof(float4)*nParticles) );
	Check( hipMalloc((void**)&d_pAccLength,sizeof(float)*nParticles) );
	Check( hipMalloc((void**)&d_cellHash,sizeof(unsigned)*nParticles) );
	Check( hipMalloc((void**)&d_cellStart,sizeof(int)*CELL_NUMX*CELL_NUMY*CELL_NUMZ) );
	Check( hipMalloc((void**)&d_pIndex,sizeof(unsigned)*nParticles) );
	Check( hipMalloc((void**)&d_simTime,sizeof(float)) );
	h_pVel = (float4 *)malloc(sizeof(float4)*nParticles);
	h_pPos = (float4 *)malloc(sizeof(float4)*nParticles);
	h_trackIndex = (unsigned *)malloc(sizeof(unsigned)*nParticles);
}

void FreeMemory(void)
{
	for(int i = 0; i<2; i++)
	{
		hipFree(d_pVel[i]);
		hipFree(d_pPos[i]);
		hipFree(d_trackIndex[i]);
	}
	hipFree(d_pNewDensity);
	hipFree(d_pBoundary);
	hipFree(d_pAcc);
	hipFree(d_pAccLength);
	hipFree(d_cellHash);
	hipFree(d_cellStart);
	hipFree(d_pIndex);
	hipFree(d_simTime);
	free(h_pVel);
	free(h_pPos);
	free(h_trackIndex);
}

int GetNumParticles(char *fname)
{
    int i = 0;
    FILE *f = fopen(fname,"r");
	
	if(f)
	{
		int x,y,z;
	  
	    while(fscanf(f,"%d,%d,%d",&x,&y,&z)==3)
	    {
			i++;
	    }
	}

	return i;
}

void Initialise(char *fname)
{
    FILE *f = fopen(fname,"r");
	
	if(f)
	{
	    int i = 0;
		int x,y,z;
	  
//	    printf("Loading...\n");
	    while(fscanf(f,"%d,%d,%d",&x,&y,&z)==3)
	    {
			if (i<nParticles)
			{
				h_pVel[i].x = 0.0f;
				h_pVel[i].y = 0.0f;
				h_pVel[i].z = 0.0f;
				h_pVel[i].w = RHO_0;
				h_pPos[i].x = x;
				h_pPos[i].y = 258-y;
				h_pPos[i].z = z;
				h_pPos[i].w = VISC_BASE;
			}
			i++;
	    }
//		printf("Loaded %d particles\n",i);
	}
}

void Display(void)
{
//	printf(":T{%f}\n",h_simTime);

	for(int i = 0; i<nParticles; i++)
	{
		//printf(":P{%d,%.2f,%.2f,%.2f,%.4g,%.4g,%.4g,%.2f,%.2f,%.2f,%.2f}\n",h_trackIndex[i],h_pPos[i].x,h_pPos[i].y,h_pPos[i].z,h_pVel[i].x,h_pVel[i].y,h_pVel[i].z,0.5f,1.0f,1.0f,1.0f);
		
		printf("%.2f,%.2f,%.2f\n",h_pPos[i].x,h_pPos[i].y,h_pPos[i].z);
	}
}

int main(int argc, char *argv[])
{
	if (argc != 2)
	{
	  printf("Usage: surfaceFlatten <input.csv>\n");
	  exit(1);
	}

	nParticles = GetNumParticles(argv[1]);
	//nParticles = 350000; // with v7_2758.csv v342500 fails, 342000 works
	
	activeArray = 0;
	int nthds = THREADS_PER_BLOCK;
	int nblks = NUM_BLOCKS;
	int nloops = 1+(nParticles-1)/(nthds*nblks);
	int nCellLoops = 1+(CELL_NUMX*CELL_NUMY*CELL_NUMZ-1)/(nthds*nblks);

	float time = 0.0f;

	
	hipSetDevice(0);

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	AllocateMemory();
	Initialise(argv[1]);

	CopyToDevice();

	InitialiseDevice<<< nblks, nthds >>>(d_pPos[activeArray],d_pAcc,d_cellHash,d_pIndex,d_trackIndex[activeArray],d_simTime,nParticles,nloops);
	thrust::sort_by_key(	thrust::device_ptr<uint>(d_cellHash),
				thrust::device_ptr<uint>(d_cellHash+nParticles),
				thrust::device_ptr<uint>(d_pIndex));
	InitCellStart<<< nblks, nthds >>>(d_cellStart,nCellLoops);
	ArrayCopy<<< nblks, nthds >>>(d_pPos[activeArray],d_pPos[1-activeArray],d_pVel[activeArray],d_pVel[1-activeArray],d_cellHash,d_cellStart,d_pIndex,nParticles,nloops);
	UpdateTrackIndex<<< nblks, nthds >>>(d_pIndex,d_trackIndex[activeArray],d_trackIndex[1-activeArray],nParticles,nloops);
	
	activeArray = 1-activeArray;

	hipEventRecord(start,0);

	int iters = 70000;

	float *maxAcc;

	for(int i = 0; i<iters;i++)
	{
			if (i && i%60 == 0)
			{
				ShepardFilter<<< nblks, nthds >>>(d_pPos[activeArray],d_pVel[activeArray],d_pNewDensity,d_cellHash,d_cellStart,nParticles,nloops);
				UpdateDensity<<< nblks, nthds >>>(d_pVel[activeArray],d_pNewDensity,nParticles,nloops);
			}

			ParticleForces<<< nblks, nthds >>>(d_pPos[activeArray],d_pVel[activeArray],d_pAcc,d_pAccLength,d_pBoundary,d_cellHash,d_cellStart,nParticles,nloops,d_trackIndex[activeArray]);
			maxAcc = thrust::raw_pointer_cast(
					thrust::max_element(thrust::device_ptr<float>(d_pAccLength),
					thrust::device_ptr<float>(d_pAccLength+nParticles))
					);
			ParticleMove<<< nblks, nthds >>>(d_pPos[activeArray],d_pVel[activeArray],d_pAcc,d_pBoundary,d_cellHash,d_pIndex,maxAcc,d_simTime,nParticles,nloops);
			thrust::sort_by_key(thrust::device_ptr<uint>(d_cellHash),
				    thrust::device_ptr<uint>(d_cellHash+nParticles),
				    thrust::device_ptr<uint>(d_pIndex));
			InitCellStart<<< nblks, nthds >>>(d_cellStart,nCellLoops);
			ArrayCopy<<< nblks, nthds >>>(d_pPos[activeArray],d_pPos[1-activeArray],d_pVel[activeArray],d_pVel[1-activeArray],d_cellHash,d_cellStart,d_pIndex,nParticles,nloops);
			UpdateTrackIndex<<< nblks, nthds >>>(d_pIndex,d_trackIndex[activeArray],d_trackIndex[1-activeArray],nParticles,nloops);

			activeArray = 1-activeArray;
	
	}
		

	CopyFromDevice();
	Display();

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time,start,stop);

	//printf("%f %d\n",time,iters);

	FreeMemory();
	return 0;
}
