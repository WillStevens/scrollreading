#include "hip/hip_runtime.h"
#include <sys/types.h>
#include <sys/socket.h>
#include <netinet/in.h>
#include <string.h>
#include <unistd.h>
#include <sys/uio.h>
#include <errno.h>
#include <stdio.h>
#include <math.h>
#include <pthread.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>

#define NUM_BLOCKS 4096
#define THREADS_PER_BLOCK 1024

#define CELL_DIMX 3.0f
#define CELL_DIMY 3.0f
#define CELL_DIMZ 3.0f

#define MAX_NEIGHBOUR_DISTANCE_SQUARED (2.1f)

#define CELL_NUMX 256
#define CELL_NUMY 256
#define CELL_NUMZ 256

#define MAXX (CELL_NUMX*CELL_DIMX)
#define MAXY (CELL_NUMY*CELL_DIMY)
#define MAXZ (CELL_NUMZ*CELL_DIMZ)

#define XZ_OFFSET 128

#define EPSILON 0.01f

#define MAXXE (MAXX-EPSILON)
#define MAXYE (MAXY-EPSILON)
#define MAXZE (MAXZ-EPSILON)

#define REPEL_FORCE_CONSTANT            0.2f
#define ATTRACT_FORCE_CONSTANT          0.4f
#define FRICTION_FORCE_CONSTANT         0.9f
#define GRAVITY_FORCE_CONSTANT 			0.0f
#define RESTORE_FORCE_CONSTANT 			0.01f

#define PI 3.14159265358979323846264f

#define RESTORE_TIME 10000 // Number of iterations after which target particles will be back to their original positions 
#define RELAX_TIME 20000 // Time allowed for hole particles to relax

#define DEBUG_OUT

__device__ float LengthVector(float3 a)
{
	return sqrtf(a.x*a.x+a.y*a.y+a.z*a.z);
}

__device__ float LengthVector(float4 a)
{
	return sqrtf(a.x*a.x+a.y*a.y+a.z*a.z);
}

__device__ float3 MultiplyVector(float3 a,float f)
{
	return make_float3(a.x*f,a.y*f,a.z*f);
}

__device__ float DotProduct(float3 a,float3 b)
{
	return a.x*b.x+a.y*b.y+a.z*b.z;
}

__global__ void InitialiseDevice(float4 *pPos, float4 *pAcc, unsigned *cellHash, unsigned *pIndex, unsigned *trackIndex, unsigned *reverseTrackIndex, int nParticles, int nloops)
{
	int thdsPerBlk = blockDim.x;	// Number of threads per block
	int blkIdx = blockIdx.x;		// Index of block
	int thdIdx = threadIdx.x;		// Index of thread within a block
	int zIdx = blkIdx*thdsPerBlk*nloops + thdIdx;
	int stepSize = blockDim.x*gridDim.x;

	for(int i = 0; i<nloops && zIdx < nParticles; i++)
	{
		cellHash[zIdx] = 
			(((int)(pPos[zIdx].x/CELL_DIMX))*CELL_NUMX +
			 ((int)(pPos[zIdx].y/CELL_DIMY)))*CELL_NUMY +
		          (int)(pPos[zIdx].z/CELL_DIMZ);
		pIndex[zIdx] = zIdx;
		trackIndex[zIdx] = zIdx;
		reverseTrackIndex[zIdx] = zIdx;
		pAcc[zIdx] = make_float4(0.0f,0.0f,0.0f,0.0f);

		zIdx += stepSize;
	}
}

__global__ void InitCellStart(int *cellStart, int nloops)
{
	int thdsPerBlk = blockDim.x;	// Number of threads per block
	int blkIdx = blockIdx.x;		// Index of block
	int thdIdx = threadIdx.x;		// Index of thread within a block
	int zIdx = blkIdx*thdsPerBlk*nloops + thdIdx;
	int stepSize = blockDim.x*gridDim.x;
	
	for(int i = 0; i<nloops && zIdx < CELL_NUMX*CELL_NUMY*CELL_NUMZ; i++)
	{
		cellStart[zIdx] = -1;

		zIdx += stepSize;
	}
}

__global__ void ArrayCopy(float4 *a0, float4 *a1, float4 *b0, float4 *b1, unsigned *cellHash, int *cellStart, unsigned *pIndex, int nParticles, int nloops)
{
	int thdsPerBlk = blockDim.x;	// Number of threads per block
	int blkIdx = blockIdx.x;		// Index of block
	int thdIdx = threadIdx.x;		// Index of thread within a block
	int zIdx = blkIdx*thdsPerBlk*nloops + thdIdx;
	int stepSize = blockDim.x*gridDim.x;

	if (zIdx==0 && 0)
	{
	  printf("pIndex...\n");
	  for(int i = 0; i<nParticles; i++)
	  {
	    printf("i=%d, pIndex[i]=%d\n",i,pIndex[i]);
	  }
	}
	
	for(int i = 0; i<nloops && zIdx < nParticles; i++)
	{
		unsigned p = pIndex[zIdx];
		a1[zIdx] = a0[p];
		b1[zIdx] = b0[p];

		if (zIdx == 0 || cellHash[zIdx] != cellHash[zIdx-1])
		{
			cellStart[cellHash[zIdx]] = zIdx;
		}

		zIdx += stepSize;
	}
}

__global__ void UpdateTrackIndex(unsigned *pIndex, unsigned *trackIndex0, unsigned *trackIndex1, unsigned *reverseTrackIndex, int nParticles, int nloops)
{
	int thdsPerBlk = blockDim.x;	// Number of threads per block
	int blkIdx = blockIdx.x;		// Index of block
	int thdIdx = threadIdx.x;		// Index of thread within a block
	int zIdx = blkIdx*thdsPerBlk*nloops + thdIdx;
	int stepSize = blockDim.x*gridDim.x;

	if (zIdx==0 && 0)
	{
	  printf("pIndex...\n");
	  for(int i = 0; i<nParticles; i++)
	  {
	    printf("i=%d, pIndex[i]=%d\n",i,pIndex[i]);
	  }
	}
	
	for(int i = 0; i<nloops && zIdx < nParticles; i++)
	{
		unsigned p = pIndex[zIdx];           // The particle that did have index p now has index zIdx
		                                     // So trackIndex0[p] is the original index of the particle that did have index p
		trackIndex1[zIdx] = trackIndex0[p];  // The particle that now has index zIdx originally had index trackIndex0[p]

		reverseTrackIndex[trackIndex0[p]] = zIdx; // reverseTrackIndex looks up current index from original index
		
		zIdx += stepSize;
	}
}

__global__ void ParticleMove(float4 *pPos, float4 *pVel, float4 *pAcc, float4 *pOriginalPos, float4 *pTargetPos, unsigned *cellHash, unsigned *trackIndex, unsigned *pIndex, int nParticles, int nloops, int nTargetParticles, int iters)
{
	int thdsPerBlk = blockDim.x;	// Number of threads per block
	int blkIdx = blockIdx.x;		// Index of block
	int thdIdx = threadIdx.x;		// Index of thread within a block
	int zIdx = blkIdx*thdsPerBlk*nloops + thdIdx;
	int stepSize = blockDim.x*gridDim.x;

	float a = RESTORE_TIME-iters;
	float b = iters;
	
	if (a<0) a=0;
	if (b>RESTORE_TIME) b=RESTORE_TIME;
	
/*
	if (zIdx==0)
	{
	  for(int i = 0; i<nParticles; i++)
	  {
        printf("xyz of %d is: %f,%f,%f\n",i,pPos[i].x,pPos[i].y,pPos[i].z);		
        printf("force on %d is: %f,%f,%f\n",i,pAcc[i].x,pAcc[i].y,pAcc[i].z);			    
	  }
	}
*/	
	for(int i = 0; i<nloops && zIdx < nParticles; i++)
	{
		unsigned ti = trackIndex[zIdx];

			if ((ti==189357 || ti==189358) && 0)
			{
				printf("%d: xyz is: %f,%f,%f\n",ti,pPos[zIdx].x,pPos[zIdx].y,pPos[zIdx].z);		
				printf("%d: force is: %f,%f,%f\n",ti,pAcc[zIdx].x,pAcc[zIdx].y,pAcc[zIdx].z);			    
			}

		pVel[zIdx].x += pAcc[zIdx].x;
		pVel[zIdx].y += pAcc[zIdx].y;
		pVel[zIdx].z += pAcc[zIdx].z;
		
    	pVel[zIdx].x *= FRICTION_FORCE_CONSTANT;
		pVel[zIdx].y *= FRICTION_FORCE_CONSTANT;
		pVel[zIdx].z *= FRICTION_FORCE_CONSTANT;

		pPos[zIdx].x += pVel[zIdx].x;
		pPos[zIdx].y += pVel[zIdx].y;
		pPos[zIdx].z += pVel[zIdx].z;

		// Apply gravitational force at this point, ready for next iteration
		// pAcc[zIdx].w is deltaDensity - set it to zero here
		pAcc[zIdx] = make_float4(0.0f,-GRAVITY_FORCE_CONSTANT,0.0f,0.0f);
		
		if (ti < nTargetParticles)
		{
		  if (0)
		  {
		    // Fix particles at a location that gradually moves towards the target
			
			pPos[zIdx] = make_float4(
	        	(pTargetPos[ti].x*b + pOriginalPos[zIdx].x*a)/RESTORE_TIME,
			    (pTargetPos[ti].y*b + pOriginalPos[zIdx].y*a)/RESTORE_TIME,
				(pTargetPos[ti].z*b + pOriginalPos[zIdx].z*a)/RESTORE_TIME,
				0);

		  }
		  
		  if (0 && ti<10)
		  {
		      printf("Tracking particle %d: xyz=%f,%f,%f target=%f,%f,%f\n",
			    ti,pPos[zIdx].x,pPos[zIdx].y,pPos[zIdx].z,pTargetPos[ti].x,pTargetPos[ti].y,pTargetPos[ti].z);
		  }
		  
		  if (1)
		  {
			// Apply a constant-magnitude force directed towards where the particle should end up...
			float3 diff = make_float3(
	        	pTargetPos[ti].x - pPos[zIdx].x,
			    pTargetPos[ti].y - pPos[zIdx].y,
				pTargetPos[ti].z - pPos[zIdx].z);

			float dist = sqrtf(diff.x*diff.x + diff.y*diff.y + diff.z*diff.z);

			if (dist > 0.1f)
			{
				float f = RESTORE_FORCE_CONSTANT / dist;
			
				pAcc[zIdx].x += f*diff.x;
				pAcc[zIdx].y += f*diff.y;
				pAcc[zIdx].z += f*diff.z;
			}
		  }
		}

		
		if (pPos[zIdx].x >= MAXXE) {pPos[zIdx].x = MAXXE; pVel[zIdx].x = 0.0f;}
		if (pPos[zIdx].y >= MAXYE) {pPos[zIdx].y = MAXYE; pVel[zIdx].y = 0.0f;}
		if (pPos[zIdx].z >= MAXZE) {pPos[zIdx].z = MAXZE; pVel[zIdx].z = 0.0f;}
		if (pPos[zIdx].x < 0.0f) {pPos[zIdx].x = 0.0f; pVel[zIdx].x = 0.0f;}
		if (pPos[zIdx].y < 0.0f) {pPos[zIdx].y = 0.0f; pVel[zIdx].y = 0.0f;}
		if (pPos[zIdx].z < 0.0f) {pPos[zIdx].z = 0.0f; pVel[zIdx].z = 0.0f;}

		cellHash[zIdx] = 
			(((int)(pPos[zIdx].x/CELL_DIMX))*CELL_NUMX +
			 ((int)(pPos[zIdx].y/CELL_DIMY)))*CELL_NUMY +
		          (int)(pPos[zIdx].z/CELL_DIMZ);

		pIndex[zIdx] = zIdx;

		zIdx += stepSize;
	}
}

#define CELLNUMAUX(x,y,z) ((x)*CELL_NUMX+(y))*CELL_NUMY+(z)
#define CELLNUM(x,y,z) ((x)<0 || (y)<0 || (z)<0 || (x)>=CELL_NUMX || (y)>=CELL_NUMY || (z)>=CELL_NUMZ)?-1:CELLNUMAUX(x,y,z)

/* Called once to count how many neighbours each particle has */
__global__ void CountNeighbours(float4 *pPos, unsigned *cellHash, int *cellStart, int *neighbourCount, unsigned *pIndex, int nParticles, int nloops)
{
	int thdsPerBlk = blockDim.x;	// Number of threads per block
	int blkIdx = blockIdx.x;		// Index of block
	int thdIdx = threadIdx.x;		// Index of thread within a block
	int zIdx = blkIdx*thdsPerBlk*nloops + thdIdx;
	int stepSize = blockDim.x*gridDim.x;

	int neighbourCell,ps;
	int cellx,celly,cellz;
	int numNeighbours;
	
	for(int i = 0; i<nloops && zIdx < nParticles; i++)
	{
	    numNeighbours = 0;
		
		cellx = (int)(pPos[zIdx].x/CELL_DIMX); 
		celly = (int)(pPos[zIdx].y/CELL_DIMY);
		cellz = (int)(pPos[zIdx].z/CELL_DIMZ);

		for(int xo=-1;xo<=1;xo++) for(int yo=-1;yo<=1;yo++) for(int zo=-1;zo<=1;zo++)
		{	
			if ((neighbourCell = CELLNUM(cellx+xo,celly+yo,cellz+zo)) != -1 &&
			    (ps = cellStart[neighbourCell]) != -1)
			{
			
				while(ps < nParticles && cellHash[ps] == neighbourCell)
				{
					if (ps != zIdx)
					{
						float3 diff = make_float3(
					        pPos[ps].x - pPos[zIdx].x,
							pPos[ps].y - pPos[zIdx].y,
							pPos[ps].z - pPos[zIdx].z);
							
						float dist2 = diff.x*diff.x+diff.y*diff.y+diff.z*diff.z;
						if (dist2 < MAX_NEIGHBOUR_DISTANCE_SQUARED)
						{
						  numNeighbours++;
						}
					}
					ps++;
				}
			}
		}

	    neighbourCount[pIndex[zIdx]] = numNeighbours;
		
		zIdx += stepSize;
	}
}

/* This is only called once */
__global__ void InitialiseNeighbours(float4 *pPos, unsigned *cellHash, int *cellStart, int *neighbourCount, uint2 *neighbourList, float *neighbourDistance, unsigned *pIndex, int nParticles, int nloops)
{
	int thdsPerBlk = blockDim.x;	// Number of threads per block
	int blkIdx = blockIdx.x;		// Index of block
	int thdIdx = threadIdx.x;		// Index of thread within a block
	int zIdx = blkIdx*thdsPerBlk*nloops + thdIdx;
	int stepSize = blockDim.x*gridDim.x;

	int neighbourCell,ps;
	int cellx,celly,cellz;
	int neighbourIndex;

/*	
	if (zIdx==0)
	{
	  printf("pIndex...\n");
	  for(int i = 0; i<nParticles; i++)
	  {
	    printf("pIndex[%d]=%d\n",i,pIndex[i]);
	  }

	  printf("neighbourCount...\n");
	  for(int i = 0; i<nParticles; i++)
	  {
	    printf("neighbourCount[%d]=%d\n",i,neighbourCount[i]);
	  }

    }
*/
	
//	if (zIdx==0) printf("Here...%d\n",pIndex[zIdx]);
	
	for(int i = 0; i<nloops && zIdx < nParticles; i++)
	{
	    neighbourIndex = 0;
		
		cellx = (int)(pPos[zIdx].x/CELL_DIMX); 
		celly = (int)(pPos[zIdx].y/CELL_DIMY);
		cellz = (int)(pPos[zIdx].z/CELL_DIMZ);

//		if (zIdx==0) printf("cell:%d,%d,%d\n",cellx,celly,cellz);

		for(int xo=-1;xo<=1;xo++) for(int yo=-1;yo<=1;yo++) for(int zo=-1;zo<=1;zo++)
		{	
			if ((neighbourCell = CELLNUM(cellx+xo,celly+yo,cellz+zo)) != -1 &&
			    (ps = cellStart[neighbourCell]) != -1)
			{
			
//		        if (zIdx==0) printf("cellx+xo,celly+yo,cellz+zo:%d,%d,%d\n",cellx+xo,celly+yo,cellz+zo);
				while(ps < nParticles && cellHash[ps] == neighbourCell)
				{
/*
					if (zIdx==0) printf("ps:%d\n",ps);
					if (zIdx==0) printf("pIndex[ps]:%d\n",pIndex[ps]);
					if (zIdx==0) printf("ps xyz:%f,%f,%f\n",pPos[ps].x,pPos[ps].y,pPos[ps].z);
					if (zIdx==0) printf("neighbourCell:%d\n",neighbourCell);
*/
					if (ps != zIdx)
					{
						float3 diff = make_float3(
					        	pPos[ps].x - pPos[zIdx].x,
							pPos[ps].y - pPos[zIdx].y,
							pPos[ps].z - pPos[zIdx].z);

//						if (zIdx==0) printf("xyz:%f,%f,%f\n",pPos[zIdx].x,pPos[zIdx].y,pPos[zIdx].z);
//						if (zIdx==0) printf("diff:%f,%f,%f\n",diff.x,diff.y,diff.z);
							
						float dist2 = diff.x*diff.x+diff.y*diff.y+diff.z*diff.z;
//						if (zIdx==0) printf("dist2:%f\n",dist2);
						if (dist2 < MAX_NEIGHBOUR_DISTANCE_SQUARED)
						{
						  // We want neighbourIndex to be the index that the particle originally have (so that we can use trackIndex)
						  // This is currently stored in pIndex, so we can use that to obtain it using trackIndex
						  int zIdxOrigIndex = pIndex[zIdx];
						  int psOrigIndex = pIndex[ps];
						  int offset = neighbourCount[zIdxOrigIndex];
/*
						  if (zIdx==0)
						  {
						    printf("zIdxOrigIndex=%d,psOrigIndex=%d,offset=%d\n",zIdxOrigIndex,psOrigIndex,offset);
						  }
*/						  
						  neighbourList[neighbourIndex+offset].x = zIdxOrigIndex;
						  neighbourList[neighbourIndex+offset].y = psOrigIndex;
						  neighbourDistance[neighbourIndex+offset] = sqrtf(dist2);
						  neighbourIndex++;
						}
					}
					ps++;
				}
			}
		}
		
		zIdx += stepSize;
	}
}

__global__ void ParticleForces(float4 *pPos, float4 *pVel, float4 *pAcc, unsigned *cellHash, int *cellStart, int nParticles, int nloops, unsigned *trackIndex)
{
	int thdsPerBlk = blockDim.x;	// Number of threads per block
	int blkIdx = blockIdx.x;		// Index of block
	int thdIdx = threadIdx.x;		// Index of thread within a block
	int zIdx = blkIdx*thdsPerBlk*nloops + thdIdx;
	int stepSize = blockDim.x*gridDim.x;

	int neighbourCell,ps;
	int cellx,celly,cellz;
	
	float4 thisAcc;

	for(int i = 0; i<nloops && zIdx < nParticles; i++)
	{
		cellx = (int)(pPos[zIdx].x/CELL_DIMX); 
		celly = (int)(pPos[zIdx].y/CELL_DIMY);
		cellz = (int)(pPos[zIdx].z/CELL_DIMZ);

		thisAcc = make_float4(0.0f,0.0f,0.0f,0.0f);

		for(int xo=-1;xo<=1;xo++) for(int yo=-1;yo<=1;yo++) for(int zo=-1;zo<=1;zo++)
		{	
			if ((neighbourCell = CELLNUM(cellx+xo,celly+yo,cellz+zo)) != -1 &&
			    (ps = cellStart[neighbourCell]) != -1)
			{
				while(ps < nParticles && cellHash[ps] == neighbourCell)
				{
					if (ps != zIdx)
					{
						float3 diff = make_float3(
					        	pPos[ps].x - pPos[zIdx].x,
							pPos[ps].y - pPos[zIdx].y,
							pPos[ps].z - pPos[zIdx].z);

							
						float dist = sqrtf(diff.x*diff.x + diff.y*diff.y + diff.z*diff.z);
						if (dist < 1.0f)
						{
							
						    float f = 1.0f-dist;

							f = f*f * REPEL_FORCE_CONSTANT / dist;

							thisAcc.x += -f * diff.x;
							thisAcc.y += -f * diff.y;
							thisAcc.z += -f * diff.z;
							
						    if (dist<0.2 && 0) // debug output for too close particles
							{
							  printf("%d,%d separated by %f (x1,y1,z1=%f,%f,%f x2,y2,z2=%f,%f,%f force=%f,%f,%f)\n",trackIndex[zIdx],trackIndex[ps],dist,pPos[zIdx].x,pPos[zIdx].y,pPos[zIdx].z,pPos[ps].x,pPos[ps].y,pPos[ps].z,-f*diff.x,-f*diff.y,-f*diff.z);
							}

						}

					}
					ps++;
				}
			}
		}

		atomicAdd(&pAcc[zIdx].x,thisAcc.x);
		atomicAdd(&pAcc[zIdx].y,thisAcc.y);
		atomicAdd(&pAcc[zIdx].z,thisAcc.z);

		zIdx += stepSize;
	}
}


__global__ void ConnectForces(float4 *pPos, float4 *pVel, float4 *pAcc, unsigned *cellHash, int *cellStart, uint2 *neighbourList, float *neighbourDistance, int nParticlePairs, int nPPloops, unsigned *reverseTrackIndex, int iters)
{
	int thdsPerBlk = blockDim.x;	// Number of threads per block
	int blkIdx = blockIdx.x;		// Index of block
	int thdIdx = threadIdx.x;		// Index of thread within a block
	int zIdx = blkIdx*thdsPerBlk*nPPloops + thdIdx;
	int stepSize = blockDim.x*gridDim.x;
	
/*
	if (zIdx == 0 && iters%1==0)
	{
	  printf("Coordinates...\n");
	  for(int i = 0; i<2; i++)
	  {
	    printf("originalIndex=%d currentIndex=%d, x=%f, y=%f, z=%f\n",i,reverseTrackIndex[i],
		  pPos[reverseTrackIndex[i]].x,
		  pPos[reverseTrackIndex[i]].y,
		  pPos[reverseTrackIndex[i]].z);
	  }
	}
*/
	for(int i = 0; i<nPPloops && zIdx < nParticlePairs; i++)
	{

		int p1,p2;
	    float distTarget;
		
		// reverseTrackIndex gives us the current index of the particle that was originally the one in the neighbourList
		p1 = reverseTrackIndex[neighbourList[zIdx].x];
		p2 = reverseTrackIndex[neighbourList[zIdx].y];
		distTarget = neighbourDistance[zIdx];
				
		
		float3 diff = make_float3(
			        	pPos[p1].x - pPos[p2].x,
						pPos[p1].y - pPos[p2].y,
						pPos[p1].z - pPos[p2].z);

		float dist = sqrtf(diff.x*diff.x + diff.y*diff.y + diff.z*diff.z);

//		if (iters%1==0) printf("Original particle pair: %d, %d\n",neighbourList[zIdx].x,neighbourList[zIdx].y);
//		if (iters%1==0) printf("Particle pair: %d, %d, distTarget:%f, dist:%f\n",p1,p2,distTarget,dist);
		
		float distFromTarget = dist - distTarget;

        float f = -distFromTarget*ATTRACT_FORCE_CONSTANT;

		float3 a = MultiplyVector(diff,f/dist);
		
//        if (iters%1==0) printf("xyz of %d is: %f,%f,%f\n",p1,pPos[p1].x,pPos[p1].y,pPos[p1].z);		
//        if (iters%1==0) printf("Force on %d is: %f,%f,%f\n",p1,a.x,a.y,a.z);		
		
		atomicAdd(&pAcc[p1].x,a.x);
		atomicAdd(&pAcc[p1].y,a.y);
		atomicAdd(&pAcc[p1].z,a.z);
	
		zIdx += stepSize;
	}
}

int nTargetParticles;
int nHoleParticles;
int nParticles;
int nParticlePairs;
float4 *d_pVel[2];
float4 *d_pPos[2];
float4 *d_pAcc;
float4 *d_pTargetPos;
float4 *d_pOriginalPos;
unsigned *d_cellHash;
int *d_cellStart;
unsigned *d_pIndex;
unsigned *d_trackIndex[2];
unsigned *d_reverseTrackIndex;
int *d_neighbourCount;       // How many neighbours does each particle have? Calculate this before allocating memory for below
uint2 *d_neighbourList;     // List of all pairs of particles p,q that are neighbours 
float *d_neighbourDistance;  // The distances between the neighbours in neighbourList


float4 *h_pVel;
float4 *h_pPos;
float4 *h_pTargetPos;
unsigned *h_trackIndex;
unsigned *h_reverseTrackIndex;
int *h_neighbourCount;
uint2 *h_neighbourList;
float *h_neighbourDistance;

int activeArray;

void Check(hipError_t e)
{
	if (e != hipSuccess)
	{
		printf("%s\n",hipGetErrorString(hipGetLastError()));
		exit(-1);
	}
}

float RandFloat(float min, float max)
{
	return min + (max-min)*((rand()%10000)/10000.0f);
}

void CopyToDevice(void)
{
	hipMemcpy(d_pVel[activeArray],h_pVel,sizeof(float4)*nParticles,hipMemcpyHostToDevice);
	hipMemcpy(d_pPos[activeArray],h_pPos,sizeof(float4)*nParticles,hipMemcpyHostToDevice);
	hipMemcpy(d_pOriginalPos,h_pPos,sizeof(float4)*nTargetParticles,hipMemcpyHostToDevice);
	hipMemcpy(d_pTargetPos,h_pTargetPos,sizeof(float4)*nTargetParticles,hipMemcpyHostToDevice);
}

void CopyFromDevice(void)
{
	hipMemcpy(h_pVel,d_pVel[activeArray],sizeof(float4)*nParticles,hipMemcpyDeviceToHost);
	hipMemcpy(h_pPos,d_pPos[activeArray],sizeof(float4)*nParticles,hipMemcpyDeviceToHost);
	hipMemcpy(h_trackIndex,d_trackIndex[activeArray],sizeof(unsigned)*nParticles,hipMemcpyDeviceToHost);
	hipMemcpy(h_reverseTrackIndex,d_reverseTrackIndex,sizeof(unsigned)*nParticles,hipMemcpyDeviceToHost);
}

void AllocateMemory(void)
{
	for(int i = 0; i<2; i++)
	{
		Check( hipMalloc((void**)&d_pVel[i],sizeof(float4)*nParticles) );
		Check( hipMalloc((void**)&d_pPos[i],sizeof(float4)*nParticles) );
		Check( hipMalloc((void**)&d_trackIndex[i],sizeof(unsigned)*nParticles) );
	}
	Check( hipMalloc((void**)&d_pTargetPos,sizeof(float4)*nTargetParticles) );
	Check( hipMalloc((void**)&d_pOriginalPos,sizeof(float4)*nTargetParticles) );
	Check( hipMalloc((void**)&d_pAcc,sizeof(float4)*nParticles) );
	Check( hipMalloc((void**)&d_reverseTrackIndex,sizeof(unsigned)*nParticles) );
	Check( hipMalloc((void**)&d_cellHash,sizeof(unsigned)*nParticles) );
	Check( hipMalloc((void**)&d_cellStart,sizeof(int)*CELL_NUMX*CELL_NUMY*CELL_NUMZ) );
	Check( hipMalloc((void**)&d_pIndex,sizeof(unsigned)*nParticles) );
	Check( hipMalloc((void**)&d_neighbourCount,sizeof(int)*nParticles) );	
	
	h_pVel = (float4 *)malloc(sizeof(float4)*nParticles);
	h_pPos = (float4 *)malloc(sizeof(float4)*nParticles);
	h_pTargetPos = (float4 *)malloc(sizeof(float4)*nTargetParticles);
	h_trackIndex = (unsigned *)malloc(sizeof(unsigned)*nParticles);
	h_reverseTrackIndex = (unsigned *)malloc(sizeof(unsigned)*nParticles);
	h_neighbourCount = (int *)malloc(sizeof(int)*nParticles);
}

int AllocateNeighbourMemory(void)
{
	int numNeighbourPairs = 0;
	
	hipMemcpy(h_neighbourCount,d_neighbourCount,sizeof(int)*nParticles,hipMemcpyDeviceToHost);

	// Count how many pairs of neighbours there are in total, and make a cumulative total so that we know the offset into
	// neighbourList and neighbourDistance to user for each particle, when building these
	
//	printf("Cumulative neighbourCount...\n");
	for(int i=0; i<nParticles; i++)
	{
	    numNeighbourPairs += h_neighbourCount[i];
	    h_neighbourCount[i] = numNeighbourPairs-h_neighbourCount[i];
//		printf("neighbourCount[%d]=%d\n",i,numNeighbourPairs);
	}

//	printf("numNeighbourParis:%d\n",numNeighbourPairs);
	
	hipMemcpy(d_neighbourCount,h_neighbourCount,sizeof(int)*nParticles,hipMemcpyHostToDevice);

	Check( hipMalloc((void**)&d_neighbourList,sizeof(uint2)*numNeighbourPairs) );	
	Check( hipMalloc((void**)&d_neighbourDistance,sizeof(float)*numNeighbourPairs) );	

	h_neighbourList = (uint2 *)malloc(sizeof(uint2)*numNeighbourPairs);
	h_neighbourDistance = (float *)malloc(sizeof(float)*numNeighbourPairs);

	return numNeighbourPairs;
}	


void FreeMemory(void)
{
	for(int i = 0; i<2; i++)
	{
		hipFree(d_pVel[i]);
		hipFree(d_pPos[i]);
		hipFree(d_trackIndex[i]);
	}
	hipFree(d_pTargetPos);
	hipFree(d_pOriginalPos);
	hipFree(d_pAcc);
	hipFree(d_reverseTrackIndex);
	hipFree(d_cellHash);
	hipFree(d_cellStart);
	hipFree(d_pIndex);
	hipFree(d_neighbourCount);
	hipFree(d_neighbourList);
	hipFree(d_neighbourDistance);
	free(h_pVel);
	free(h_pPos);
	free(h_pTargetPos);
	free(h_trackIndex);
	free(h_reverseTrackIndex);
	free(h_neighbourCount);
	free(h_neighbourList);
	free(h_neighbourDistance);
}

int GetNumParticles(char *fname)
{
    int i = 0;
    FILE *f = fopen(fname,"r");
	
	if(f)
	{
		float x,y,z;
	  
	    while(fscanf(f,"%f,%f,%f",&x,&y,&z)==3)
	    {
			i++;
	    }
		
		fclose(f);
	}

	return i;
}

// Two vectors that define the projection plane
// The third vector in this array is the normal to the projection plane, calculated from the other two
float planeVectors[3][3] = 
	{ { 1, 0, 0 },
	  { 0, 0, 1 } };

float projectN(float x, float y, float z, int n)
{
	float r = (x*planeVectors[n][0]+y*planeVectors[n][1]+z*planeVectors[n][2])/1000.0f;
	
	return r;
}

float inverseRotate[3][3];
float inverseTranslate[3];

float inverseProjectN(float x, float y, float z, int n)
{
	float r = (x*inverseRotate[n][0]+y*inverseRotate[n][1]+z*inverseRotate[n][2])/1000.0f;
	
	return r;
}

void InitialiseInverse(void)
{
//  printf("Initialising inverse\n");
  
  // computes the inverse of a matrix m
  double det = planeVectors[0][0] * (planeVectors[1][1] * planeVectors[2][2] - planeVectors[2][1] * planeVectors[1][2]) -
               planeVectors[0][1] * (planeVectors[1][0] * planeVectors[2][2] - planeVectors[1][2] * planeVectors[2][0]) +
               planeVectors[0][2] * (planeVectors[1][0] * planeVectors[2][1] - planeVectors[1][1] * planeVectors[2][0]);

  double invdet = 1000000.0f / det;

  inverseRotate[0][ 0] = (planeVectors[1][ 1] * planeVectors[2][ 2] - planeVectors[2][ 1] * planeVectors[1][ 2]) * invdet;
  inverseRotate[0][ 1] = (planeVectors[0][ 2] * planeVectors[2][ 1] - planeVectors[0][ 1] * planeVectors[2][ 2]) * invdet;
  inverseRotate[0][ 2] = (planeVectors[0][ 1] * planeVectors[1][ 2] - planeVectors[0][ 2] * planeVectors[1][ 1]) * invdet;
  inverseRotate[1][ 0] = (planeVectors[1][ 2] * planeVectors[2][ 0] - planeVectors[1][ 0] * planeVectors[2][ 2]) * invdet;
  inverseRotate[1][ 1] = (planeVectors[0][ 0] * planeVectors[2][ 2] - planeVectors[0][ 2] * planeVectors[2][ 0]) * invdet;
  inverseRotate[1][ 2] = (planeVectors[1][ 0] * planeVectors[0][ 2] - planeVectors[0][ 0] * planeVectors[1][ 2]) * invdet;
  inverseRotate[2][ 0] = (planeVectors[1][ 0] * planeVectors[2][ 1] - planeVectors[2][ 0] * planeVectors[1][ 1]) * invdet;
  inverseRotate[2][ 1] = (planeVectors[2][ 0] * planeVectors[0][ 1] - planeVectors[0][ 0] * planeVectors[2][ 1]) * invdet;
  inverseRotate[2][ 2] = (planeVectors[0][ 0] * planeVectors[1][ 1] - planeVectors[1][ 0] * planeVectors[0][ 1]) * invdet;	   

//  printf("Done initialising inverse\n");
  
}

void Initialise(char *target, char *flat, char *holes)
{
    FILE *f = fopen(target,"r");
	
	if(f)
	{
	    int i = 0;
		float x,y,z;
		float minx=1000000.0f,miny=1000000.0f,minz=1000000.0f;
	  
//	    printf("Loading...\n");
	    while(fscanf(f,"%f,%f,%f",&x,&y,&z)==3)
	    {
			if (i<nParticles-nHoleParticles)
			{
				h_pTargetPos[i].x = projectN(x,y,z,0);
				h_pTargetPos[i].y = projectN(x,y,z,2);
				h_pTargetPos[i].z = projectN(x,y,z,1);
				
				if (h_pTargetPos[i].x < minx) minx = h_pTargetPos[i].x;
				if (h_pTargetPos[i].y < miny) miny = h_pTargetPos[i].y;
				if (h_pTargetPos[i].z < minz) minz = h_pTargetPos[i].z;
			}
			i++;
	    }
		
		inverseTranslate[0] = minx - XZ_OFFSET;
		inverseTranslate[1] = miny;
		inverseTranslate[2] = minz - XZ_OFFSET;
		
		for(i = 0; i<nParticles-nHoleParticles; i++)
		{
		   h_pTargetPos[i].x = h_pTargetPos[i].x - minx + XZ_OFFSET;
		   h_pTargetPos[i].y -= miny;
		   h_pTargetPos[i].z = h_pTargetPos[i].z - minz + XZ_OFFSET;
		}
		
//		printf("Loaded %d target particles\n",i);
		
		fclose(f);
	}

    f = fopen(flat,"r");

	if(f)
	{
	    int i = 0;
		float x,y,z;
		  
	    while(fscanf(f,"%f,%f,%f",&x,&y,&z)==3)
	    {
			if (i<nParticles)
			{
				h_pVel[i].x = 0.0f;
				h_pVel[i].y = 0.0f;
				h_pVel[i].z = 0.0f;
				h_pPos[i].x = x;
				h_pPos[i].y = y;
				h_pPos[i].z = z;
			}
			i++;
	    }
		
//		printf("Loaded %d flat particles\n",i);
		
		fclose(f);
	}

    f = fopen(holes,"r");
	
	if(f)
	{
	    int i = 0;
		float x,y,z;
	  
//	    printf("Loading...\n");
	    while(fscanf(f,"%f,%f,%f",&x,&y,&z)==3)
	    {
			if (i<nHoleParticles)
			{
				h_pVel[i+nTargetParticles].x = 0.0f;
				h_pVel[i+nTargetParticles].y = 0.0f;
				h_pVel[i+nTargetParticles].z = 0.0f;
				h_pPos[i+nTargetParticles].x = x;
				h_pPos[i+nTargetParticles].y = y;
				h_pPos[i+nTargetParticles].z = z;
			}
			i++;
	    }
		
//		printf("Loaded %d hole particles\n",i);
		
		fclose(f);
	}
}

void Display(void)
{
//	printf(":T{%f}\n",h_simTime);
	float x,y,z;
	
    float maxy = 0.0f;
	
	for(int i = 0; i<nParticles; i++)
	{
	  if (h_pPos[i].y>maxy) maxy = h_pPos[i].y;
    }
	
//	printf("maxy=%f\n",maxy);
	
	
	for(int i = 0; i<nParticles; i++)
	{
		//printf(":P{%d,%.2f,%.2f,%.2f,%.4g,%.4g,%.4g,%.2f,%.2f,%.2f,%.2f}\n",h_trackIndex[i],h_pPos[i].x,h_pPos[i].y,h_pPos[i].z,h_pVel[i].x,h_pVel[i].y,h_pVel[i].z,0.5f,1.0f,1.0f,1.0f);

		// Output in the same order as the input particles
		
		int currentIndex = h_reverseTrackIndex[i];

		x = h_pPos[currentIndex].x;
		y = h_pPos[currentIndex].y;
		z = h_pPos[currentIndex].z;


		if (0 && i==0)
		  printf("0-pos: %.2f,%.2f,%.2f\n",x,y,z);
		
		x += inverseTranslate[0];
		y += inverseTranslate[1];
		z += inverseTranslate[2];

		if (0 && i==0)
		  printf("0-translated: %.2f,%.2f,%.2f\n",x,y,z);

		
		float xf = inverseProjectN(x,z,y,0);
		float yf = inverseProjectN(x,z,y,1);
		float zf = inverseProjectN(x,z,y,2);
		
		printf("%.2f,%.2f,%.2f\n",xf,yf,zf);
	}
	
}

int main(int argc, char *argv[])
{
	if (argc != 10)
	{
	  printf("Usage: surfaceUnFlatten <input.csv> <flat.csv> <holes.csv> x1 y1 z1 x2 y2 z2\n");
	  exit(1);
	}

	for(int i = 0; i<2; i++)
	  for(int j = 0; j<3; j++)
	  {
	    planeVectors[i][j] = atof(argv[j+i*3+argc-6]);
//		printf("%f\n",planeVectors[i][j]);
	  }
	  
//    printf("Projection plane and normal vectors normalised to length 1000:\n");
    for(int i = 0; i<3; i++)
    {
	  if (i==2)
	  {
	    /* Cross product of the two plane vectors gives the normal vector */
	    planeVectors[2][0] = (planeVectors[0][1]*planeVectors[1][2]-planeVectors[0][2]*planeVectors[1][1])/1000.0f;
	    planeVectors[2][1] = (planeVectors[0][2]*planeVectors[1][0]-planeVectors[0][0]*planeVectors[1][2])/1000.0f;
	    planeVectors[2][2] = (planeVectors[0][0]*planeVectors[1][1]-planeVectors[0][1]*planeVectors[1][0])/1000.0f;
      }
	
  	  float magnitude = sqrtf(planeVectors[i][0]*planeVectors[i][0]+planeVectors[i][1]*planeVectors[i][1]+planeVectors[i][2]*planeVectors[i][2]);
		
	  for(int j = 0; j<3; j++)
	  {
		planeVectors[i][j] /= (magnitude/1000.0f);
		
//		  printf("%f ",planeVectors[i][j]);
	  }
		
//	  printf("\n");
    }

	InitialiseInverse();
/*
	for(int i = 0; i<3; i++)
	  for(int j = 0; j<3; j++)
	  {
		printf("%f\n",inverseRotate[i][j]);
	  }
*/	
	nTargetParticles = GetNumParticles(argv[1]);
	nHoleParticles = GetNumParticles(argv[3]);
	nParticles = GetNumParticles(argv[2])+ nHoleParticles;
	
	if (nParticles != nTargetParticles+nHoleParticles)
	{
	  printf("Particle count mismatch\n");
	  exit(2);
	}
	
	activeArray = 0;
	int nthds = THREADS_PER_BLOCK;
	int nblks = NUM_BLOCKS;
	int nloops = 1+(nParticles-1)/(nthds*nblks);
	int nCellLoops = 1+(CELL_NUMX*CELL_NUMY*CELL_NUMZ-1)/(nthds*nblks);

	float time = 0.0f;

	
	hipSetDevice(0);

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

//	printf("About to allocate memory\n");
	AllocateMemory();
	Initialise(argv[1],argv[2],argv[3]);
	
/*
	printf("Inverse translation:\n");
	for(int i = 0; i<3; i++)
	{
      printf("%f\n",inverseTranslate[i]);
	}
*/	
	CopyToDevice();

	InitialiseDevice<<< nblks, nthds >>>(d_pPos[activeArray],d_pAcc,d_cellHash,d_pIndex,d_trackIndex[activeArray],d_reverseTrackIndex,nParticles,nloops);
	thrust::sort_by_key(	thrust::device_ptr<uint>(d_cellHash),
				thrust::device_ptr<uint>(d_cellHash+nParticles),
				thrust::device_ptr<uint>(d_pIndex));

								
	InitCellStart<<< nblks, nthds >>>(d_cellStart,nCellLoops);
	
	ArrayCopy<<< nblks, nthds >>>(d_pPos[activeArray],d_pPos[1-activeArray],d_pVel[activeArray],d_pVel[1-activeArray],d_cellHash,d_cellStart,d_pIndex,nParticles,nloops);

	UpdateTrackIndex<<< nblks, nthds >>>(d_pIndex,d_trackIndex[activeArray],d_trackIndex[1-activeArray],d_reverseTrackIndex,nParticles,nloops);
	
	activeArray = 1-activeArray;
 
    CountNeighbours<<< nblks, nthds >>>(d_pPos[activeArray],d_cellHash,d_cellStart,d_neighbourCount, d_pIndex,nParticles, nloops);

//	printf("About to allocate neighbour memory\n");
	nParticlePairs = AllocateNeighbourMemory();

	int nPPloops = 1+(nParticlePairs-1)/(nthds*nblks);

    InitialiseNeighbours<<< nblks, nthds >>>(d_pPos[activeArray],d_cellHash,d_cellStart,d_neighbourCount, d_neighbourList, d_neighbourDistance, d_pIndex,nParticles, nloops);

	hipMemcpy(h_neighbourList,d_neighbourList,sizeof(uint2)*nParticlePairs,hipMemcpyDeviceToHost);
	hipMemcpy(h_neighbourDistance,d_neighbourDistance,sizeof(float)*nParticlePairs,hipMemcpyDeviceToHost);

/*
	printf("NeighbourList...\n");
	for(int i = 0; i<nParticlePairs; i++)
	{
	  printf("n1=%d,n2=%d,dist=%f\n",h_neighbourList[i].x,h_neighbourList[i].y,h_neighbourDistance[i]);
	}
*/

	hipEventRecord(start,0);

	int iters = RESTORE_TIME+RELAX_TIME;

	for(int i = 0; i<iters;i++)
	{
//	        printf("Iteration %d\n",i);

			ParticleForces<<< nblks, nthds >>>(d_pPos[activeArray],d_pVel[activeArray],d_pAcc,d_cellHash,d_cellStart,nParticles,nloops,d_trackIndex[activeArray]);

			ConnectForces<<< nblks, nthds >>>(d_pPos[activeArray],d_pVel[activeArray],d_pAcc,d_cellHash,d_cellStart,d_neighbourList,d_neighbourDistance,nParticlePairs,nPPloops,d_reverseTrackIndex,i);

			ParticleMove<<< nblks, nthds >>>(d_pPos[activeArray],d_pVel[activeArray],d_pAcc,d_pOriginalPos,d_pTargetPos,d_cellHash,d_trackIndex[activeArray]
			,d_pIndex,nParticles,nloops,nTargetParticles,i);
			thrust::sort_by_key(thrust::device_ptr<uint>(d_cellHash),
				    thrust::device_ptr<uint>(d_cellHash+nParticles),
				    thrust::device_ptr<uint>(d_pIndex));
			InitCellStart<<< nblks, nthds >>>(d_cellStart,nCellLoops);
			ArrayCopy<<< nblks, nthds >>>(d_pPos[activeArray],d_pPos[1-activeArray],d_pVel[activeArray],d_pVel[1-activeArray],d_cellHash,d_cellStart,d_pIndex,nParticles,nloops);
			UpdateTrackIndex<<< nblks, nthds >>>(d_pIndex,d_trackIndex[activeArray],d_trackIndex[1-activeArray],d_reverseTrackIndex,nParticles,nloops);

			activeArray = 1-activeArray;	
	}
		

	CopyFromDevice();
	Display();
	
//    for(int i = 0; i<200; i++)
//	  printf("%d\n",h_neighbourIndex[i]);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time,start,stop);

	//printf("%f %d\n",time,iters);

	FreeMemory();
	return 0;
}
